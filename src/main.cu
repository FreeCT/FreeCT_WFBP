#include "hip/hip_runtime.h"
/* CTBangBang is GPU and CPU CT reconstruction Software */
/* Copyright (C) 2015  John Hoffman */

/* This program is free software; you can redistribute it and/or */
/* modify it under the terms of the GNU General Public License */
/* as published by the Free Software Foundation; either version 2 */
/* of the License, or (at your option) any later version. */

/* This program is distributed in the hope that it will be useful, */
/* but WITHOUT ANY WARRANTY; without even the implied warranty of */
/* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the */
/* GNU General Public License for more details. */

/* You should have received a copy of the GNU General Public License */
/* along with this program; if not, write to the Free Software */
/* Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA. */

/* Questions and comments should be directed to */
/* jmhoffman@mednet.ucla.edu with "CTBANGBANG" in the subject line*/


#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <regex.h>
#include <cstdarg>
#include <unistd.h>
#include <sys/types.h>
#include <pwd.h>

#include <recon_structs.h>
#include <setup.h>
#include <rebin_filter.h>
#include <rebin_filter_cpu.h>
#include <backproject.h>
#include <backproject_cpu.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
	{
	    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
	    if (abort) exit(code);
	}
}

void log(int verbosity, const char *string, ...);

void usage(){
    printf("\n");
    printf("usage: recon [options] input_prm_file\n\n");
    printf("    Options:\n");
    printf("          -v: verbose.\n");
    printf("          -t: test files will be written to desktop.\n");
    printf("    --no-gpu: run program exclusively on CPU. Will override --device=i option.\n");
    printf("  --device=i: run on GPU device number 'i'\n");
    printf("    --timing: Display timing information for each step of the recon process\n");
    printf("\n");
    printf("Copyright John Hoffman 2015\n\n");
    exit(0);
}


int main(int argc, char ** argv){
    
    struct recon_metadata mr;
    memset(&mr,0,sizeof(struct recon_metadata));

    // Parse any command line arguments
    if (argc<2)
	usage();
    
    regex_t regex_dev;
    regmatch_t regmatch_dev;
    if (regcomp(&regex_dev,"--device=*",0)!=0){
	printf("Regex didn't work properly\n");
	exit(1);
    }
    
    for (int i=1;i<(argc-1);i++){
	if (strcmp(argv[i],"-t")==0){
	    mr.flags.testing=1;
	}
	else if (strcmp(argv[i],"-v")==0){
	    mr.flags.verbose=1;
	}
	else if (strcmp(argv[i],"--no-gpu")==0){
	    mr.flags.no_gpu=1;
	}
	else if (regexec(&regex_dev,argv[i],1,&regmatch_dev,0)==0){
	    mr.flags.set_device=1;
	    sscanf(argv[i],"--device=%d",&mr.flags.device_number);
	}
	else if (strcmp(argv[i],"--timing")==0){
	    mr.flags.timing=1;
	}
	else{
	    usage();
	}
    }

    /* --- Get working directory and User's home directory --- */
    struct passwd *pw=getpwuid(getuid());
    const char * homedir=pw->pw_dir;
    strcpy(mr.homedir,homedir);
    getcwd(mr.install_dir,4096*sizeof(char));
    
    /* --- Step 0: configure our processor (CPU or GPU) */
    // We want to send to the GPU furthest back in the list which is
    // unlikely to have a display connected.  We also check for the
    // user passing a specific device number via the command line

    int device_count=0;
    hipGetDeviceCount(&device_count);
    if (device_count==0){
	mr.flags.no_gpu=1;
    }

    hipEvent_t start,stop;
    
    if (mr.flags.no_gpu==0){
	int device;
	if (mr.flags.set_device==1){
	    log(mr.flags.verbose,"CUDA device %d requested.\n",mr.flags.device_number);
	    log(mr.flags.verbose,"Attempting to set device... ");
	    hipSetDevice(mr.flags.device_number);
	    hipGetDevice(&device);
	    if (device!=mr.flags.device_number){
		printf("There was a problem setting device.\n");
	    }
	    else{
		log(mr.flags.verbose,"success!\n");
	    }
	}
	else{
	    hipSetDevice(device_count-1);
	    hipGetDevice(&device);
	}	
	log(mr.flags.verbose,"Working on GPU %i \n",device);
	hipDeviceReset();
    }
    else{
	log(mr.flags.verbose,"Working on CPU\n");
    }
    
    
    /* --- Step 1-3 handled by functions in setup.cu --- */
    // Step 1: Parse input file
    log(mr.flags.verbose,"Reading PRM file...\n");
    mr.rp=configure_recon_params(argv[argc-1]);

    // Step 2a: Setup scanner geometry
    log(mr.flags.verbose,"Configuring scanner geometry...\n");
    mr.cg=configure_ct_geom(&mr);
    
    // Step 2b: Configure all remaining information
    log(mr.flags.verbose,"Configuring final reconstruction parameters...\n");
    configure_reconstruction(&mr);

    for (int i=0;i<mr.ri.n_blocks;i++){

	update_block_info(&mr);
	
	// Step 3: Extract raw data from file into memory
	log(mr.flags.verbose,"Reading raw data from file...\n");
	extract_projections(&mr);
    
	/* --- Step 4 handled by functions in rebin_filter.cu --- */
	// Step 4: Rebin and filter
	log(mr.flags.verbose,"Rebinning and filtering data...\n");

	if (mr.flags.timing){
	    hipEventCreate(&start);
	    hipEventCreate(&stop);
	    hipEventRecord(start);
	}

	if (mr.flags.no_gpu==1)
	    rebin_filter_cpu(&mr);
	else
	    rebin_filter(&mr);

	if (mr.flags.timing){
	    hipEventRecord(stop);
	    hipEventSynchronize(stop);
	    float milli=0.0f;
	    hipEventElapsedTime(&milli,start,stop);
	    printf("%.2f ms to rebin & filter\n",milli);
	    hipEventDestroy(start);
	    hipEventDestroy(stop);
	}
	
	/* --- Step 5 handled by functions in backproject.cu ---*/
	// Step 5: Backproject
	log(mr.flags.verbose,"Backprojecting...\n");

	if (mr.flags.timing){
	    hipEventCreate(&start);
	    hipEventCreate(&stop);
	    hipEventRecord(start);
	}
    
	if (mr.flags.no_gpu==1)
	    backproject_cpu(&mr);
	else
	    backproject(&mr);

	if (mr.flags.timing){
	    hipEventRecord(stop);
	    hipEventSynchronize(stop);
	    float milli=0.0f;
	    hipEventElapsedTime(&milli,start,stop);
	    printf("%.2f ms to backproject\n",milli);
	    hipEventDestroy(start);
	    hipEventDestroy(stop);
	}
	
    }
    // Step 6: Save image data to disk (found in setup.cu)
    log(mr.flags.verbose,"Writing image data to disk...\n");
    finish_and_cleanup(&mr);

    log(mr.flags.verbose,"Done.\n");

    hipDeviceReset();
    return 0;
   
}

void log(int verbosity, const char *string,...){
    va_list args;
    va_start(args,string);

    if (verbosity){
	vprintf(string,args);
	va_end(args);
    } 
}
