#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <recon_structs.h>
#include <setup.h>
#include <rebin_filter.h>
#include <rebin_filter_cpu.h>
#include <backproject.h>
#include <backproject_cpu.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
	{
	    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
	    if (abort) exit(code);
	}
}

void usage(){
    printf("\n");
    printf("usage: recon [options] input_prm_file\n\n");
    printf("    Options:\n");
    printf("          -v: verbose.\n");
    printf("          -t: test files will be written to desktop.\n");
    printf("\n");
    printf("Copyright John Hoffman 2015\n\n");
    exit(0);
}


int main(int argc, char ** argv){
    
    struct recon_metadata mr;
    memset(&mr,0,sizeof(struct recon_metadata));

    // Parse any command line arguments
    if (argc<2)
	usage();
    
    for (int i=1;i<(argc-1);i++){
	if (strcmp(argv[i],"-t")==0){
	    mr.flags.testing=1;
	}
	else if (strcmp(argv[i],"-v")==0){
	    mr.flags.verbose=1;
	}
	else if (strcmp(argv[i],"--no-gpu")==0){
	    mr.flags.no_gpu=1;
	}
	else{
	    usage();
	}
    }

    /* --- Step 0: configure our GPU */
    // We want to send to the GPU furthest back in the list
    // which is unlikely to have a display connected
    int device_count=0;
    hipGetDeviceCount(&device_count);
    if (device_count==0){
	mr.flags.no_gpu=1;
    }
    
    if (mr.flags.verbose){
	if (mr.flags.no_gpu==0)
	    printf("Working on GPU %i \n",device_count-1);
	else
	    printf("Working on CPU\n");
    }
    gpuErrchk(hipSetDevice(device_count-1));
    hipDeviceReset();
    
    /* --- Step 1-3 handled by functions in setup.cu --- */
    // Step 1: Parse input file
    if (mr.flags.verbose)
	printf("Reading PRM file...\n");
    mr.rp=configure_recon_params(argv[argc-1]);

    // Step 2a: Setup scanner geometry
    if (mr.flags.verbose)
	printf("Configuring scanner geometry...\n");
    mr.cg=configure_ct_geom(mr.rp);
    
    // Step 2b: Configure all remaining information
    if (mr.flags.verbose)
	printf("Configuring final reconstruction parameters...\n");
    configure_reconstruction(&mr);

    for (int i=0;i<mr.ri.n_blocks;i++){

	update_block_info(&mr);
	
	// Step 3: Extract raw data from file into memory
	if (mr.flags.verbose)
	    printf("Reading raw data from file...\n");
	extract_projections(&mr);
    
	/* --- Step 4 handled by functions in rebin_filter.cu --- */
	// Step 4: Rebin and filter
	if (mr.flags.verbose)
	    printf("Rebinning and filtering data...\n");

	if (mr.flags.no_gpu==1)
	    rebin_filter_cpu(&mr);
	else
	    rebin_filter(&mr);
	
	/* --- Step 5 handled by functions in backproject.cu ---*/
	// Step 5: Backproject
	if (mr.flags.verbose)
	    printf("Backprojecting...\n");

	if (mr.flags.no_gpu==1)
	    backproject_cpu(&mr);
	else
	    backproject(&mr);
	
    }
    // Step 6: Save image data to disk (found in setup.cu)
    if (mr.flags.verbose)
	printf("Writing image data to disk...\n");
    finish_and_cleanup(&mr);

    if (mr.flags.verbose)
	printf("Done.\n");

    hipDeviceReset();
    return 0;
   
}
