#include "hip/hip_runtime.h"
/* CTBangBang is GPU and CPU CT reconstruction Software */
/* Copyright (C) 2015  John Hoffman */

/* This program is free software; you can redistribute it and/or */
/* modify it under the terms of the GNU General Public License */
/* as published by the Free Software Foundation; either version 2 */
/* of the License, or (at your option) any later version. */

/* This program is distributed in the hope that it will be useful, */
/* but WITHOUT ANY WARRANTY; without even the implied warranty of */
/* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the */
/* GNU General Public License for more details. */

/* You should have received a copy of the GNU General Public License */
/* along with this program; if not, write to the Free Software */
/* Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA. */

/* Questions and comments should be directed to */
/* jmhoffman@mednet.ucla.edu with "CTBANGBANG" in the subject line*/


#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <regex.h>
#include <cstdarg>
#include <unistd.h>
#include <sys/types.h>
#include <pwd.h>

#include <recon_structs.h>
#include <setup.h>
#include <rebin_filter.h>
#include <rebin_filter_cpu.h>
#include <backproject.h>
#include <backproject_cpu.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
	{
	    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
	    if (abort) exit(code);
	}
}

void log(int verbosity, const char *string, ...);
void split_path_file(char** p, char** f, char *pf);

void usage(){
    printf("\n");
    printf("usage: recon [options] input_prm_file\n\n");
    printf("    Options:\n");
    printf("          -v: verbose.\n");
    printf("          -t: test files will be written to desktop.\n");
    printf("    --no-gpu: run program exclusively on CPU. Will override --device=i option.\n");
    printf("  --device=i: run on GPU device number 'i'\n");
    printf("    --timing: Display timing information for each step of the recon process\n");
    printf(" --benchmark: Writes timing data to file used by benchmarking tool\n");    
    printf("\n");
    printf("Copyright John Hoffman 2015\n\n");
    exit(0);
}


int main(int argc, char ** argv){

    struct recon_metadata mr;
    memset(&mr,0,sizeof(struct recon_metadata));

    // Parse any command line arguments
    if (argc<2)
	usage();
    
    regex_t regex_dev;
    regmatch_t regmatch_dev;
    if (regcomp(&regex_dev,"--device=*",0)!=0){
	printf("Regex didn't work properly\n");
	exit(1);
    }
    
    for (int i=1;i<(argc-1);i++){
	if (strcmp(argv[i],"-t")==0){
	    mr.flags.testing=1;
	}
	else if (strcmp(argv[i],"-v")==0){
	    mr.flags.verbose=1;
	}
	else if (strcmp(argv[i],"--no-gpu")==0){
	    mr.flags.no_gpu=1;
	}
	else if (regexec(&regex_dev,argv[i],1,&regmatch_dev,0)==0){
	    mr.flags.set_device=1;
	    sscanf(argv[i],"--device=%d",&mr.flags.device_number);
	}
	else if (strcmp(argv[i],"--timing")==0){
	    mr.flags.timing=1;
	}
	else if (strcmp(argv[i],"--benchmark")==0){
	    mr.flags.benchmark=1;
	} 
	else{
	    usage();
	}
    }

    log(mr.flags.verbose,"\n-------------------------\n"
                         "|      CTBangBang       |\n"
                         "-------------------------\n\n");

    log(mr.flags.verbose,"CHECKING INPUT PARAMETERS AND CONFIGURING RECONSTRUCTION\n"
	                 "\n");
    
    /* --- Get working directory and User's home directory --- */
    struct passwd *pw=getpwuid(getuid());
    
    const char * homedir=pw->pw_dir;
    strcpy(mr.homedir,homedir);
    char full_exe_path[4096]={0};
    char * exe_path=(char*)calloc(4096,sizeof(char));
    char * exe_name=(char*)calloc(255,sizeof(char));
    readlink("/proc/self/exe",full_exe_path,4096);
    split_path_file(&exe_path,&exe_name,full_exe_path);
    strcpy(mr.install_dir,exe_path);
    mr.install_dir[strlen(mr.install_dir)-1]=0;
    
    /* --- Step 0: configure our processor (CPU or GPU) */
    // We want to send to the GPU furthest back in the list which is
    // unlikely to have a display connected.  We also check for the
    // user passing a specific device number via the command line

    int device_count=0;
    hipGetDeviceCount(&device_count);
    if (device_count==0){
	mr.flags.no_gpu=1;
    }

    // Configure the GPU/CPU selection
    if (mr.flags.no_gpu==0){
	int device;
	if (mr.flags.set_device==1){
	    log(mr.flags.verbose,"CUDA device %d requested.\n",mr.flags.device_number);
	    log(mr.flags.verbose,"Attempting to set device... ");
	    hipSetDevice(mr.flags.device_number);
	    hipGetDevice(&device);
	    if (device!=mr.flags.device_number){
		printf("There was a problem setting device.\n");
	    }
	    else{
		log(mr.flags.verbose,"success!\n");
	    }
	}
	else{
	    hipSetDevice(device_count-1);
	    hipGetDevice(&device);
	}	
	log(mr.flags.verbose,"Working on GPU %i \n",device);
	hipDeviceReset();
    }
    else{
	log(mr.flags.verbose,"Working on CPU\n");
    }

    // --timing cuda events
    hipEvent_t start,stop;

    // Set up benchmarking variables and output file if requested
    char fullpath[4096+255];
    strcpy(fullpath,mr.homedir);
    strcat(fullpath,"/Desktop/.tmp_benchmark.bin");
    FILE * benchmark_file;
    if (mr.flags.benchmark){
	benchmark_file=fopen(fullpath,"a");
	fseek(benchmark_file,0,SEEK_END);
    }

    hipEvent_t bench_master_start,bench_master_stop,bench_start,bench_stop;
    if (mr.flags.benchmark){
	hipEventCreate(&bench_master_start);
	hipEventCreate(&bench_master_stop);
	hipEventRecord(bench_master_start);
    }
    
    /* --- Step 1-3 handled by functions in setup.cu --- */
    // Step 1: Parse input file
    log(mr.flags.verbose,"Reading PRM file...\n");
    mr.rp=configure_recon_params(argv[argc-1]);

    // Step 2a: Setup scanner geometry
    log(mr.flags.verbose,"Configuring scanner geometry...\n");
    mr.cg=configure_ct_geom(&mr);
    
    // Step 2b: Configure all remaining information
    log(mr.flags.verbose,"Configuring final reconstruction parameters...\n");
    configure_reconstruction(&mr);

    log(mr.flags.verbose,"Allowed recon range: %.2f to %.2f\n",mr.ri.allowed_begin,mr.ri.allowed_end);

    log(mr.flags.verbose,"\nSTARTING RECONSTRUCTION\n\n");
    
    for (int i=0;i<mr.ri.n_blocks;i++){

	update_block_info(&mr);
	
	log(mr.flags.verbose,"----------------------------\n"
                             "Working on block %d of %d \n",i+1,mr.ri.n_blocks);
	
	// Step 3: Extract raw data from file into memory
	log(mr.flags.verbose,"Reading raw data from file...\n");
	extract_projections(&mr);
    
	/* --- Step 4 handled by functions in rebin_filter.cu --- */
	// Step 4: Rebin and filter
	log(mr.flags.verbose,"Rebinning and filtering data...\n");

	if (mr.flags.timing){
	    hipEventCreate(&start);
	    hipEventCreate(&stop);
	    hipEventRecord(start);
	}

	if (mr.flags.benchmark){
	    hipEventCreate(&bench_start);
	    hipEventCreate(&bench_stop);
	    hipEventRecord(bench_start);
	}

	if (mr.flags.no_gpu==1)
	    rebin_filter_cpu(&mr);
	else
	    rebin_filter(&mr);

	if (mr.flags.timing){
	    hipEventRecord(stop);
	    hipEventSynchronize(stop);
	    float milli=0.0f;
	    hipEventElapsedTime(&milli,start,stop);
	    printf("%.2f ms to rebin & filter\n",milli);
	    hipEventDestroy(start);
	    hipEventDestroy(stop);
	}
	if (mr.flags.benchmark){
	    hipEventRecord(bench_stop);
	    hipEventSynchronize(bench_stop);
	    float milli=0.0f;
	    hipEventElapsedTime(&milli,bench_start,bench_stop);
	    // write the benchmark data to file
	    fwrite(&milli,sizeof(float),1,benchmark_file);
	    hipEventDestroy(bench_start);
	    hipEventDestroy(bench_stop);
	}

	/* --- Step 5 handled by functions in backproject.cu ---*/
	// Step 5: Backproject
	log(mr.flags.verbose,"Backprojecting...\n");

	if (mr.flags.timing){
	    hipEventCreate(&start);
	    hipEventCreate(&stop);
	    hipEventRecord(start);
	}

	if (mr.flags.benchmark){
	    hipEventCreate(&bench_start);
	    hipEventCreate(&bench_stop);
	    hipEventRecord(bench_start);
	}

	if (mr.flags.no_gpu==1)
	    backproject_cpu(&mr);
	else
	    backproject(&mr);

	if (mr.flags.timing){
	    hipEventRecord(stop);
	    hipEventSynchronize(stop);
	    float milli=0.0f;
	    hipEventElapsedTime(&milli,start,stop);
	    printf("%.2f ms to backproject\n",milli);
	    hipEventDestroy(start);
	    hipEventDestroy(stop);
	}
	
	if (mr.flags.benchmark){
	    hipEventRecord(bench_stop);
	    hipEventSynchronize(bench_stop);
	    float milli=0.0f;
	    hipEventElapsedTime(&milli,bench_start,bench_stop);
	    // write the benchmark data to file
	    fwrite(&milli,sizeof(float),1,benchmark_file);
	    hipEventDestroy(bench_start);
	    hipEventDestroy(bench_stop);
	}

	
    }


    // Step 6: Save image data to disk (found in setup.cu)
    log(mr.flags.verbose,"----------------------------\n\n");
    log(mr.flags.verbose,"Writing image data to %s/Desktop/%s.img\n",mr.homedir,mr.rp.raw_data_file);
    finish_and_cleanup(&mr);

    log(mr.flags.verbose,"Done.\n");

    if (mr.flags.benchmark){
	hipEventRecord(bench_master_stop);
	hipEventSynchronize(bench_master_stop);
	float milli=0.0f;
	hipEventElapsedTime(&milli,bench_master_start,bench_master_stop);
	// write the benchmark data to file
	fwrite(&milli,sizeof(float),1,benchmark_file);
	hipEventDestroy(bench_master_start);
	hipEventDestroy(bench_master_stop);
	fclose(benchmark_file);
    }

    hipDeviceReset();
    return 0;
   
}

void log(int verbosity, const char *string,...){
    va_list args;
    va_start(args,string);

    if (verbosity){
	vprintf(string,args);
	va_end(args);
    } 
}

void split_path_file(char** p, char** f, char *pf) {
    char *slash = pf, *next;
    while ((next = strpbrk(slash + 1, "\\/"))) slash = next;
    if (pf != slash) slash++;
    *p = strndup(pf, slash - pf);
    *f = strdup(slash);
}
